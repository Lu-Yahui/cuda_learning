#include "hip/hip_runtime.h"
#include "vector_addition.h"

__global__ void VecAddKernel(float* A, float* B, float* C, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

/**
 * @brief CPU version of vector addition
 *
 * @param h_A
 * @param h_B
 * @param h_C
 * @param n
 */
void VecAddCpu(float* h_A, float* h_B, float* h_C, int n) {
  for (int i = 0; i < n; ++i) {
    h_C[i] = h_A[i] + h_B[i];
  }
}

/**
 * @brief GPU version of vector addition
 *
 * @param h_A
 * @param h_B
 * @param h_C
 * @param n
 */
void VecAddGpu(float* h_A, float* h_B, float* h_C, int n) {
  int size = n * sizeof(float);

  // allocate device memory
  float* d_A;
  float* d_B;
  float* d_C;
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // copy host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // kernel launch
  int block_size = static_cast<int>(std::ceil(n / 256.0));
  VecAddKernel<<<block_size, 256>>>(d_A, d_B, d_C, n);

  // copy result from device memory to host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // release cuda memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
