#include "hip/hip_runtime.h"
#include "matrix_mul.h"

__global__ void SimpleMatrixMulKernel(float* M, float* N, float* P, int width) {
  int row = blockDim.y * blockIdx.y + threadIdx.y;
  int col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row < width && col < width) {
    float value = 0.0F;
    for (int k = 0; k < width; ++k) {
      value += M[row * width + k] * N[k * width + col];
    }

    P[row * width + col] = value;
  }
}

void SimpleMatrixMulGpu(float* M, float* N, float* P, int width) {
  // allocate device memory
  float* d_M;
  float* d_N;
  float* d_P;
  int size = sizeof(float) * width * width;
  hipMalloc((void**)&d_M, size);
  hipMalloc((void**)&d_N, size);
  hipMalloc((void**)&d_P, size);

  // copy data from host memory to device memory
  hipMemcpy(d_M, M, size, hipMemcpyHostToDevice);
  hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);

  // setup grid dim
  int thread_x_per_block = 32;
  int thread_y_per_block = 32;
  int block_x_per_grid = std::ceil(width / static_cast<double>(thread_x_per_block));
  int block_y_per_grid = std::ceil(width / static_cast<double>(thread_y_per_block));
  dim3 dim_grid(block_x_per_grid, block_y_per_grid, 1);
  dim3 dim_block(thread_x_per_block, thread_y_per_block, 1);

  // launch kernel
  SimpleMatrixMulKernel<<<dim_grid, dim_block>>>(d_M, d_N, d_P, width);

  // copy result back to host memory
  hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);

  // release cuda memory
  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);
}
